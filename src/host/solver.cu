#include "hip/hip_runtime.h"
#include <unordered_map>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "solver.h"
#include "../device/solver_kernels.h"
#include "../wordle.h"

using namespace std;

int num_color_perm(int word_len)
{
  int base = 1;
  for (int i = 0; i < word_len; i++)
    base *= 3;
  return base;
}

float calculate_expected_information(vector<int> &word, vector<vector<int>> &dictionary)
{
  unordered_map<int, int> colorings;
  for (int i = 0; i < dictionary.size(); i++)
  {
    vector<int> current_word = dictionary[i];
    int coloring = Wordle::generate_coloring(word, current_word);
    if (colorings.count(coloring))
    {
      colorings[coloring]++;
    }
    else
    {
      colorings[coloring] = 1;
    }
  }

  float expected_information = 0.0f;
  for (auto it = colorings.begin(); it != colorings.end(); it++)
  {
    int key = it->first;
    int occurances = it->second;
    float p = float(occurances) / dictionary.size();
    if (p > 0)
    {
      expected_information += p * log2(1 / p);
    }
  }
  return expected_information;
}

void update_dictionary(vector<int> &guess, vector<vector<int>> &dictionary, int color)
{
  int old_dict_size = dictionary.size();
  for (auto it = dictionary.begin(); it != dictionary.end();)
  {
    int coloring = Wordle::generate_coloring(*it, guess);
    if (coloring != color)
    {
      it = dictionary.erase(it);
    }
    else
    {
      it++;
    }
  }
  float p = float(dictionary.size()) / old_dict_size;
  cout << "Actual Information: " << log2(1 / p) << endl;
}

vector<int> Solver::serial_solver(vector<vector<int>> guesses, vector<int> colors)
{
  if (guesses.size() > 0)
  {
    update_dictionary(guesses.back(), dictionary, colors.back());
  }
  vector<int> best_guess = {};
  float highest_expected_information = -1;
  for (int i = 0; i < dictionary.size(); i++)
  {
    vector<int> current_word = dictionary[i];
    float expected_information = calculate_expected_information(current_word, dictionary);
    if (expected_information > highest_expected_information)
    {
      highest_expected_information = expected_information;
      best_guess = current_word;
    }
  }
  cout << "Expected Information: " << highest_expected_information << endl;
  return best_guess;
}

vector<int> Solver::cuda_solver(vector<vector<int>> guesses, vector<int> colors)
{
  cout << "Starting CUDA Solver" << endl;
  if (guesses.size() > 0)
  {
    update_dictionary(guesses.back(), dictionary, colors.back());
  }
  int num_words = dictionary.size();
  int *dictionary_arr = new int[num_words * word_len];
  for (int i = 0; i < num_words * word_len; i++)
    dictionary_arr[i] = dictionary[i / word_len][i % word_len];

  int color_perm = num_color_perm(word_len);

  float *information = new float[num_words];
  int *colorings = new int[num_words * color_perm];
  for (int i = 0; i < num_words * color_perm; i++)
    colorings[i] = 0;
  int *_dictionary;
  float *_information;
  int *_colorings;
  hipMalloc((void **)&_dictionary, num_words * word_len * sizeof(int));
  hipMalloc((void **)&_information, num_words * sizeof(float));
  hipMalloc((void **)&_colorings, num_words * color_perm * sizeof(int));

  hipMemcpy(_colorings, colorings, num_words * color_perm * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(_dictionary, dictionary_arr, num_words * word_len * sizeof(int), hipMemcpyHostToDevice);
  calculate_expected_information_cuda(num_words, word_len, color_perm, _dictionary, _information, _colorings);
  // calculate_expected_information_cuda_shmem_full(num_words, word_len, color_perm, _dictionary, _information, _colorings);

  hipMemcpy(information, _information, num_words * sizeof(float), hipMemcpyDeviceToHost);

  vector<int> best_guess = {};
  float highest_expected_information = -1;
  for (int i = 0; i < dictionary.size(); i++)
  {
    vector<int> current_word = dictionary[i];
    float expected_information = information[i];
    if (expected_information > highest_expected_information)
    {
      highest_expected_information = expected_information;
      best_guess = current_word;
    }
  }

  hipFree(_dictionary);
  hipFree(_information);
  hipFree(_colorings);
  cout << "Expected Information: " << highest_expected_information << endl;
  return best_guess;
}