#include <unordered_map>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "solver.h"
#include "../device/solver_kernels.h"
#include "../wordle.h"

using namespace std;

vector<int> Solver::solve(vector<vector<int>> guesses, vector<int> colors)
{
  cout << "Starting CUDA Solver" << endl;
  if (guesses.size() > 0)
  {
    update_dictionary(guesses.back(), dictionary, colors.back());
  }
  int num_words = dictionary.size();
  int *dictionary_arr = new int[num_words * word_len];
  for (int i = 0; i < num_words * word_len; i++)
    dictionary_arr[i] = dictionary[i / word_len][i % word_len];

  int color_perm = num_color_perm(word_len);

  float *information = new float[num_words];
  int *colorings = new int[num_words * color_perm];
  for (int i = 0; i < num_words * color_perm; i++)
    colorings[i] = 0;
  int *_dictionary;
  float *_information;
  int *_colorings;
  hipMalloc((void **)&_dictionary, num_words * word_len * sizeof(int));
  hipMalloc((void **)&_information, num_words * sizeof(float));
  hipMalloc((void **)&_colorings, num_words * color_perm * sizeof(int));

  hipMemcpy(_colorings, colorings, num_words * color_perm * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(_dictionary, dictionary_arr, num_words * word_len * sizeof(int), hipMemcpyHostToDevice);
  calculate_expected_information_cuda(num_words, word_len, color_perm, _dictionary, _information, _colorings);
  // calculate_expected_information_cuda_shmem_full(num_words, word_len, color_perm, _dictionary, _information, _colorings);

  hipMemcpy(information, _information, num_words * sizeof(float), hipMemcpyDeviceToHost);

  vector<int> best_guess = {};
  float highest_expected_information = -1;
  for (int i = 0; i < dictionary.size(); i++)
  {
    vector<int> current_word = dictionary[i];
    float expected_information = information[i];
    if (expected_information > highest_expected_information)
    {
      highest_expected_information = expected_information;
      best_guess = current_word;
    }
  }

  hipFree(_dictionary);
  hipFree(_information);
  hipFree(_colorings);
  cout << "Expected Information: " << highest_expected_information << endl;
  return best_guess;
}