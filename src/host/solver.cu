#include "hip/hip_runtime.h"
#include <unordered_map>
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "solver.h"
#include "../device/solver_kernels.h"
#include "../wordle.h"

using namespace std;


int num_color_perm(int word_len)
{
  int base = 1;
  for (int i = 0; i < word_len; i++)
    base *= 3;
  return base;
}

vector<int> calculate_expected_information(vector<int> &word, vector<vector<int>> &dictionary)
{
  vector<int> colorings(59049);
  for (int i = 0; i < dictionary.size(); i++)
  {
    vector<int> current_word = dictionary[i];
    int coloring = Wordle::generate_coloring(word, current_word);
    colorings[coloring]++;
  }

  // float expected_information = 0.0f;
  // for (int occurances : colorings)
  // {
  //   float p = float(occurances) / dictionary.size();
  //   if (p > 0)
  //   {
  //     expected_information += p * log2(1 / p);
  //   }
  // }
  return colorings;
}

void Solver::update_dictionary(vector<int> guess, int color)
{
  int old_dict_size = dictionary.size();
  for (auto it = dictionary.begin(); it != dictionary.end();)
  {
    int coloring = Wordle::generate_coloring(*it, guess);
    if (coloring != color)
    {
      it = dictionary.erase(it);
    }
    else
    {
      it++;
    }
  }
  float p = float(dictionary.size()) / old_dict_size;
  cout << "Old Dict Size: " << old_dict_size << " New Dict Size: " << dictionary.size() << endl;
  cout << "Actual Information: " << log2(1 / p) << endl;
}

vector<pair<float, pair<vector<int>, vector<int>>>> Solver::serial_solver(vector<vector<int>> guesses, vector<int> colors)
{
  vector<pair<float, pair<vector<int>, vector<int>>>> info = {};
  vector<int> best_guess = {};
  float highest_expected_information = -1;
  for (int i = 0; i < dictionary.size(); i++)
  {
    vector<int> current_word = dictionary[i];
    // cout << "calculating expected info" << endl;
    vector<int> colorings = calculate_expected_information(current_word, dictionary);

    float expected_information = 0.0f;
    for (int occurances : colorings)
    {
      float p = float(occurances) / dictionary.size();
      if (p > 0)
      {
        expected_information += p * log2(1 / p);
      }
    }

    if (expected_information > highest_expected_information)
    {
      highest_expected_information = expected_information;
      best_guess = current_word;
    }
    info.push_back({expected_information, {current_word, colorings}});
  }

  sort(info.begin(), info.end());
  return info;
}

vector<pair<float, vector<int>>> Solver::cuda_solver(vector<vector<int>> guesses, vector<int> colors, bool shmem, bool multi_color)
{

  int num_words = dictionary.size();
  int *dictionary_arr = new int[num_words * word_len];
  for (int i = 0; i < num_words * word_len; i++)
    dictionary_arr[i] = dictionary[i / word_len][i % word_len];

  int color_perm = num_color_perm(word_len);

  float *information = new float[num_words];
  int *colorings = new int[num_words * color_perm];
  for (int i = 0; i < num_words * color_perm; i++)
    colorings[i] = 0;
  int *_dictionary;
  float *_information;
  int *_colorings;
  hipMalloc((void **)&_dictionary, num_words * word_len * sizeof(int));
  hipMalloc((void **)&_information, num_words * sizeof(float));
  hipMalloc((void **)&_colorings, num_words * color_perm * sizeof(int));

  hipMemcpy(_colorings, colorings, num_words * color_perm * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(_dictionary, dictionary_arr, num_words * word_len * sizeof(int), hipMemcpyHostToDevice);

  if (shmem) {
    calculate_expected_information_cuda_shmem_full(num_words, word_len, _dictionary, _information);
  } else if (multi_color) {
    calculate_expected_information_cuda_percolor(num_words, word_len, _dictionary, _information);
  } else {
    calculate_expected_information_cuda(num_words, word_len, _dictionary, _information);
  }
  
  hipMemcpy(information, _information, num_words * sizeof(float), hipMemcpyDeviceToHost);

  vector<pair<float, vector<int>>> info = {};
  vector<int> best_guess = {};
  float highest_expected_information = -1;
  for (int i = 0; i < dictionary.size(); i++)
  {
    vector<int> current_word = dictionary[i];
    float expected_information = information[i];
    if (expected_information > highest_expected_information)
    {
      highest_expected_information = expected_information;
      best_guess = current_word;
    }
    info.push_back({expected_information, current_word});
  }

  hipFree(_dictionary);
  hipFree(_information);
  hipFree(_colorings);
  cout << "Expected Information: " << highest_expected_information << endl;
  
  sort(info.begin(), info.end());
  return info;
}