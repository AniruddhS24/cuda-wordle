#include "hip/hip_runtime.h"
#include "solver_kernels.h"
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

#define BLOCK_SIZE 32

__global__ void calculate_expected_information_kernel(int num_words, int word_len, int *dictionary, float *information)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < num_words)
    {
        int colorings[243]; // num_colorings (3^word_len)
        int word_st = tid * word_len;

        for (int i = 0; i < num_words; i++)
        {
            int cur_word_st = i * word_len;

            int coloring[5];   // word_len
            int letter_ct[27]; // vocab_size

            // compute coloring
            for (int j = 0; j < word_len; j++)
                coloring[j] = 0;
            for (int j = 0; j < 27; j++)
                letter_ct[j] = 0;
            for (int j = 0; j < word_len; j++)
                letter_ct[dictionary[word_st + j]]++;
            for (int j = 0; j < word_len; j++)
            {
                int cur = dictionary[cur_word_st + j];
                if (cur == dictionary[word_st + j])
                {
                    coloring[j] = 1;
                    letter_ct[cur]--;
                }
            }

            for (int j = 0; j < word_len; j++)
            {
                int cur = dictionary[cur_word_st + j];
                if (coloring[j] == 1)
                    continue;
                if (letter_ct[cur] > 0)
                {
                    coloring[j] = 2;
                    letter_ct[cur]--;
                }
                else
                {
                    coloring[j] = 0;
                }
            }

            // convert coloring to base 3
            int base = 1;
            int c = 0;
            for (int j = 0; j < 5; j++)
            {
                c += base * coloring[j];
                base *= 3;
            }

            // if (tid == 1)
            //     printf("tid %d Coloring: %d\n", tid, c);

            // increment coloring
            colorings[c]++;
        }

        float expected_info = 0.0;
        for (int i = 0; i < 243; i++)
        {
            float p = (float)colorings[i] / (float)num_words;
            if (p > 0)
                expected_info += p * log2(1 / p);
        }
        information[tid] = expected_info;
    }
}

void calculate_expected_information_cuda(int num_words, int word_len, int *dictionary, float *information)
{
    dim3 blockGrid((num_words + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 threadBlock(BLOCK_SIZE);
    calculate_expected_information_kernel<<<blockGrid, threadBlock>>>(num_words, word_len, dictionary, information);
}